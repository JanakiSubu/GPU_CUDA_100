// Compile with: nvcc vec_cublas.cu -o vec_cublas -lstdc++ -lcublas


#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main() {
    const int N = 10;
    float A[N], B[N], C[N];

    // Initialize input vectors A and B with sample values
    for(int i = 0; i < N; i++) {
        A[i] = i;
        B[i] = i;
    }

    // Create cuBLAS context handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Allocate device memory for vectors A and B
    float *d_a, *d_b;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_b, N * sizeof(float));

    // Transfer vectors A and B from host to device
    hipMemcpy(d_a, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Set the scalar multiplier for the AXPY operation
    const float alpha = 1.0f;

    // Perform vector addition on GPU: d_b = alpha * d_a + d_b
    hipblasSaxpy(handle, N, &alpha, d_a, 1, d_b, 1);

    // Copy result vector from device to host (result is in d_b)
    hipMemcpy(C, d_b, N * sizeof(float), hipMemcpyDeviceToHost);

    // Display the result vector C
    for(int i = 0; i < N; i++) {
        std::cout << C[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory and destroy cuBLAS handle
    hipFree(d_a);
    hipFree(d_b);
    hipblasDestroy(handle);

    return 0;
}
